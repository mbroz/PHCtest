#include "hip/hip_runtime.h"
/**
 * A simple implementation of Blake2b's internal permutation 
 * in the form of a sponge.  Experimental CUDA implementation.
 * 
 * Note: Implemented without shared memory optimizations.
 * 
 * Author: The Lyra PHC team (http://www.lyra-kdf.net/) -- 2014.
 * 
 * This software is hereby placed in the public domain.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <string.h>
#include <stdio.h>
#include <time.h>
#include "Sponge.h"
#include "Lyra2.h"

/**
 * Initializes the Sponge State. The first 512 bits are set to zeros and the remainder 
 * receive Blake2b's IV as per Blake2b's specification. <b>Note:</b> Even though sponges
 * typically have their internal state initialized with zeros, Blake2b's G function
 * has a fixed point: if the internal state and message are both filled with zeros. the 
 * resulting permutation will always be a block filled with zeros; this happens because 
 * Blake2b does not use the constants originally employed in Blake2 inside its G function, 
 * relying on the IV for avoiding possible fixed points.
 * 
 * @param state         The 1024-bit array to be initialized
 */
void initState(uint64_t state[/*16*/]){
    hipMemset(state, 0,            64);  //first 512 bis are zeros
    if ( hipSuccess != hipGetLastError() ) {
        printf( "CUDA memory setting error in file %s, line %d!\n",  __FILE__, __LINE__  );
            printf( "Error: %s \n", hipGetErrorString(hipGetLastError()) );
            exit(EXIT_FAILURE);
    }

    uint64_t *state2 = &state[8];
    hipMemcpy(state2, blake2b_IV,   64, hipMemcpyHostToDevice);
    if ( hipSuccess != hipGetLastError() ) {
        printf( "CUDA memory copy error in file %s, line %d!\n",  __FILE__, __LINE__  );
            printf( "Error: %s \n", hipGetErrorString(hipGetLastError()) );
            exit(EXIT_FAILURE);
    }
}

/**
 * Execute Blake2b's G function, with all 12 rounds.
 * 
 * @param v     A uint64_t array to be processed by Blake2b's G function
 */
__device__ static void blake2bLyra2(uint64_t *v) {
    ROUND_LYRA(0);
    ROUND_LYRA(1);
    ROUND_LYRA(2);
    ROUND_LYRA(3);
    ROUND_LYRA(4);
    ROUND_LYRA(5);
    ROUND_LYRA(6);
    ROUND_LYRA(7);
    ROUND_LYRA(8);
    ROUND_LYRA(9);
    ROUND_LYRA(10);
    ROUND_LYRA(11);
}

/**
*	Wrapper to call from CPU.
*/
__global__ static void blake2bLyra(uint64_t *v) {
    blake2bLyra2(v);
}



/**
 * Executes a reduced version of Blake2b's G function with only one round
 * @param v     A uint64_t array to be processed by Blake2b's G function
 */
__device__ static void reducedBlake2bLyra2(uint64_t *v) {
    ROUND_LYRA(0);
}


/**
 * Performs a squeeze operation, using Blake2b's G function as the 
 * internal permutation
 * 
 * @param state      The current state of the sponge 
 * @param out        Array that will receive the data squeezed
 * @param len        The number of bytes to be squeezed into the "out" array
 */
void squeeze(uint64_t *state, byte *out, unsigned int len) {
    int fullBlocks = len / BLOCK_LEN_BYTES;
    hipError_t  erro;
    int i;
    //Squeezes full blocks
    for (i = 0; i < fullBlocks; i++) {
        hipMemcpy(out + (BLOCK_LEN_BYTES * i), state, BLOCK_LEN_BYTES, hipMemcpyDeviceToHost);
	
		erro = hipGetLastError();
		if ( hipSuccess !=  erro ) {
			printf( "Error in file %s, line %d!\n", __FILE__, __LINE__ );
			printf( "Error: %s \n", hipGetErrorString(erro) );
		}

		blake2bLyra<<<1,1>>>(state);
		erro = hipGetLastError();
		if ( hipSuccess != erro ) {
			printf( "Error in file %s, line %d!\n", __FILE__, __LINE__ );
			printf( "Error: %s \n", hipGetErrorString(erro) );
		}
    }
    
    //Squeezes remaining bytes
    hipMemcpy(out + (BLOCK_LEN_BYTES * fullBlocks), state, (len % BLOCK_LEN_BYTES), hipMemcpyDeviceToHost);
    if ( hipSuccess != hipGetLastError() ) {
		printf( "Erro no arquivo %s, na linha %d!\n", __FILE__, __LINE__ );
    }
}



/**
 * Performs an absorb operation for a single block (BLOCK_LEN_INT64 words
 * of type uint64_t), using Blake2b's G function as the internal permutation
 * 
 * @param state The current state of the sponge 
 * @param in    The block to be absorbed (BLOCK_LEN_INT64 words)
 */
__device__ void absorbBlock2(uint64_t *state, const uint64_t *in) {
    //XORs the first BLOCK_LEN_INT64 words of "in" with the current state
    state[0] ^= in[0];
    state[1] ^= in[1];
    state[2] ^= in[2];
    state[3] ^= in[3];
    state[4] ^= in[4];
    state[5] ^= in[5];
    state[6] ^= in[6];
    state[7] ^= in[7];
    state[8] ^= in[8];
    state[9] ^= in[9];
    state[10] ^= in[10];
    state[11] ^= in[11];
	
    //Applies the transformation f to the sponge's state	
    blake2bLyra2(state);
}

/**
*	Wrapper to call from CPU.
*/
__global__ void absorbBlock(uint64_t *state, const uint64_t *in) {
    absorbBlock2(state, in);
}



/** 
 * Performs a squeeze operation for two rows in sequence, using 
 * reduced Blake2b's G function as the internal permutation
 * 
 * @param state     The current state of the sponge 
 * @param row0      Row to receive the data squeezed
 */
__global__ void reducedSqueezeRow(uint64_t* state, uint64_t* row0) {  // Já convertido
    uint64_t* ptr64 = row0;     // Pointer to position to be filled first (M[0])
    int i;
    for (i = 0; i < N_COLS; i++) {
        ptr64[0] = state[0];
        ptr64[1] = state[1];
        ptr64[2] = state[2];
        ptr64[3] = state[3];
        ptr64[4] = state[4];
        ptr64[5] = state[5];
        ptr64[6] = state[6];
        ptr64[7] = state[7];
        ptr64[8] = state[8];
        ptr64[9] = state[9];
        ptr64[10] = state[10];
        ptr64[11] = state[11];	
		
        //Goes to next block (column) that will receive the squeezed data		
        ptr64 += BLOCK_LEN_INT64;
		
        //Applies the reduced-round transformation f to the sponge's state        
		reducedBlake2bLyra2(state);
    }
}



/**
 * Performs a duplex operation over "M[rowInOut] XOR M[rowIn]", writing the output "rand"
 * on M[rowOut] and making "M[rowInOut] =  M[rowInOut] XOR rotW(rand)", where rotW is a 64-bit 
 * rotation to the left.
 *
 * @param state          The current state of the sponge 
 * @param rowIn          Row used only as input
 * @param rowInOut       Row used as input and to receive output after rotation
 * @param rowOut         Row receiving the output
 *
 */
__device__ void reducedDuplexRowSetup2(uint64_t *state, uint64_t *rowIn, uint64_t *rowInOut, uint64_t *rowOut){
    uint64_t* ptr64In = rowIn; 			//In Lyra2: pointer to prev
    uint64_t* ptr64InOut = rowInOut; 	//In Lyra2: pointer to row*
    uint64_t* ptr64Out = rowOut; 		//In Lyra2: pointer to row
    int i;
    for (i = 0; i < N_COLS; i++){
         //Absorbing "M[rowInOut] XOR M[rowIn]"
        state[0] ^= ptr64InOut[0] ^ ptr64In[0];
        state[1] ^= ptr64InOut[1] ^ ptr64In[1];
        state[2] ^= ptr64InOut[2] ^ ptr64In[2];
        state[3] ^= ptr64InOut[3] ^ ptr64In[3];
        state[4] ^= ptr64InOut[4] ^ ptr64In[4];
        state[5] ^= ptr64InOut[5] ^ ptr64In[5];
        state[6] ^= ptr64InOut[6] ^ ptr64In[6];
        state[7] ^= ptr64InOut[7] ^ ptr64In[7];
        state[8] ^= ptr64InOut[8] ^ ptr64In[8];
        state[9] ^= ptr64InOut[9] ^ ptr64In[9];
        state[10] ^= ptr64InOut[10] ^ ptr64In[10];
        state[11] ^= ptr64InOut[11] ^ ptr64In[11];
        
        //Applies the reduced-round transformation f to the sponge's state
        reducedBlake2bLyra2(state);

        //M[rowOut][col] = rand
        ptr64Out[0] = state[0];
        ptr64Out[1] = state[1];
        ptr64Out[2] = state[2];
        ptr64Out[3] = state[3];
        ptr64Out[4] = state[4];
        ptr64Out[5] = state[5];
        ptr64Out[6] = state[6];
        ptr64Out[7] = state[7];
        ptr64Out[8] = state[8];
        ptr64Out[9] = state[9];
        ptr64Out[10] = state[10];
        ptr64Out[11] = state[11];

        //M[rowInOut][col] = M[rowInOut][col] XOR rotW(rand)
        ptr64InOut[0] ^= state[11];
        ptr64InOut[1] ^= state[0];
        ptr64InOut[2] ^= state[1];
        ptr64InOut[3] ^= state[2];
        ptr64InOut[4] ^= state[3];
        ptr64InOut[5] ^= state[4];
        ptr64InOut[6] ^= state[5];
        ptr64InOut[7] ^= state[6];
        ptr64InOut[8] ^= state[7];
        ptr64InOut[9] ^= state[8];
        ptr64InOut[10] ^= state[9];
        ptr64InOut[11] ^= state[10];
		
        //Goes to next column (i.e., next block in sequence)
        ptr64InOut += BLOCK_LEN_INT64;
        ptr64In += BLOCK_LEN_INT64;
        ptr64Out += BLOCK_LEN_INT64;

    }
}

/**
*	Wrapper to call from CPU.
*/
__global__ void reducedDuplexRowSetup(uint64_t *state, uint64_t *rowa, uint64_t *prev, uint64_t *newRow){
    reducedDuplexRowSetup2(state, rowa, prev, newRow);
}



/**
 * Performs a duplex operation over "M[rowInOut] XOR M[rowIn]", using the output "rand"
 * to make "M[rowOut][col] = M[rowOut][col] XOR rand" and "M[rowInOut] = M[rowInOut] XOR rotW(rand)", 
 * where rotW is a 64-bit rotation to the left.
 *
 * @param state          The current state of the sponge 
 * @param rowIn          Row used only as input
 * @param rowInOut       Row used as input and to receive output after rotation
 * @param rowOut         Row receiving the output
 *
 */
__device__ void reducedDuplexRow2(uint64_t *state, uint64_t *rowIn, uint64_t *rowInOut, uint64_t *rowOut) {
    uint64_t* ptr64InOut = rowInOut; 	//In Lyra2: pointer to row*
    uint64_t* ptr64In = rowIn;          //In Lyra2: pointer to prev
    uint64_t* ptr64Out = rowOut; 		//In Lyra2: pointer to row
    int i;
    for (i = 0; i < N_COLS; i++) {
	
	//Absorbing "M[rowInOut] XOR M[rowIn]"
        state[0] ^= ptr64InOut[0] ^ ptr64In[0];
        state[1] ^= ptr64InOut[1] ^ ptr64In[1];
        state[2] ^= ptr64InOut[2] ^ ptr64In[2];
        state[3] ^= ptr64InOut[3] ^ ptr64In[3];
        state[4] ^= ptr64InOut[4] ^ ptr64In[4];
        state[5] ^= ptr64InOut[5] ^ ptr64In[5];
        state[6] ^= ptr64InOut[6] ^ ptr64In[6];
        state[7] ^= ptr64InOut[7] ^ ptr64In[7];
        state[8] ^= ptr64InOut[8] ^ ptr64In[8];
        state[9] ^= ptr64InOut[9] ^ ptr64In[9];
        state[10] ^= ptr64InOut[10] ^ ptr64In[10];
        state[11] ^= ptr64InOut[11] ^ ptr64In[11];
        
        //Applies the reduced-round transformation f to the sponge's state
        reducedBlake2bLyra2(state);

	//M[rowOut][col] = M[rowOut][col] XOR rand
        ptr64Out[0] ^= state[0];
        ptr64Out[1] ^= state[1];
        ptr64Out[2] ^= state[2];
        ptr64Out[3] ^= state[3];
        ptr64Out[4] ^= state[4];
        ptr64Out[5] ^= state[5];
        ptr64Out[6] ^= state[6];
        ptr64Out[7] ^= state[7];
        ptr64Out[8] ^= state[8];
        ptr64Out[9] ^= state[9];
        ptr64Out[10] ^= state[10];
        ptr64Out[11] ^= state[11];

	//M[rowInOut][col] = M[rowInOut][col] XOR rotW(rand)
        ptr64InOut[0] ^= state[11];
        ptr64InOut[1] ^= state[0];
        ptr64InOut[2] ^= state[1];
        ptr64InOut[3] ^= state[2];
        ptr64InOut[4] ^= state[3];
        ptr64InOut[5] ^= state[4];
        ptr64InOut[6] ^= state[5];
        ptr64InOut[7] ^= state[6];
        ptr64InOut[8] ^= state[7];
        ptr64InOut[9] ^= state[8];
        ptr64InOut[10] ^= state[9];
        ptr64InOut[11] ^= state[10];

        //Goes to next column (i.e., next block in sequence)
        ptr64Out += BLOCK_LEN_INT64;
        ptr64InOut += BLOCK_LEN_INT64;
        ptr64In += BLOCK_LEN_INT64;
    } 
}
 
/**
*	Wrapper to call from CPU.
*/ 
__global__ void reducedDuplexRow(uint64_t *state, uint64_t *prev, uint64_t *rowa, uint64_t *row) {
    reducedDuplexRow2(state, prev, rowa, row);
}


//====================== Setup Phase =====================//
__global__ void setupGPU(uint64_t *state, uint64_t *MemMatrix, int nRows){
    int rowa = 0;
    int row  = 2;
    int prev = 1;

    uint64_t*  ptr64a;
    uint64_t*  ptr64p;
    uint64_t*  ptr64n;

    do{

        ptr64p = &MemMatrix[(prev * ROW_LEN_INT64)];   // 0
        ptr64a = &MemMatrix[(rowa * ROW_LEN_INT64)];   // 1
        ptr64n = &MemMatrix[(row  * ROW_LEN_INT64)];   // 2

        reducedDuplexRowSetup2(state, ptr64p, ptr64a, ptr64n);

        //updates the value of row* (deterministically picked during Setup))
        rowa = rowa - 1;
        if(rowa < 0){
            rowa = prev;
        }
        //update prev: it now points to the last row ever computed
        prev = row;
        //updates row: does to the next row to be computed
        row = row + 1;
    } while (row < nRows );
}


//================== Wandering Phase =====================//  (stateDev, MemMatrixDev, timeCost, nRows, rowADev);
__global__ void wandering(uint64_t *state, uint64_t *MemMatrix, int timeCost, int nRows, int *rowA){
    int maxIndex = nRows - 1;
    int rowa = 0;		 	//index of row* (a previous row, deterministically picked during Setup and randomly picked during Wandering)
    int row = maxIndex;   	//index of row to be processed 
    int prev = 0;			//index of prev (last row ever computed/modified)
    int tau;				//Time Loop interator
    uint64_t * MemMatrixDev_P;
    uint64_t * MemMatrixDev_A;   
    uint64_t * MemMatrixDev_R;   

    for (tau = 1; tau <= timeCost; tau++){
        
        //========= Iterations for an odd tau  ==========
        row = maxIndex; //Odd iterations of the Wandering phase start with the last row ever computed
        prev = 0;       //The companion "prev" is 0
		
        do{
            //Selects a pseudorandom index row*
            //rowa = ((unsigned int)state[0] ^ prev) & maxIndex; //(USE THIS IF nRows IS A POWER OF 2)
            rowa = ((unsigned int)state[0] ^ prev) % nRows;		//(USE THIS FOR THE "GENERIC" CASE)

            MemMatrixDev_P = &MemMatrix[(prev * ROW_LEN_INT64)];
            MemMatrixDev_A = &MemMatrix[(rowa * ROW_LEN_INT64)];   
            MemMatrixDev_R = &MemMatrix[(row  * ROW_LEN_INT64)];   

            //Performs a reduced-round duplexing operation over M[row*] XOR M[prev], updating both M[row*] and M[row]
            reducedDuplexRow2(state, MemMatrixDev_P , MemMatrixDev_A, MemMatrixDev_R);

            prev = row;
            row = row - 1;            
        } while (row >= 0);

        if (++tau > timeCost) {
            break; //end of the Wandering phase
        }
		
		//========= Iterations for an even tau  ==========
        row = 0;            //Even iterations of the Wandering phase start with row = 0
        prev = maxIndex;    //The companion "prev" is the last row in the memory matrix
        do {
            //rowa = ((unsigned int)state[0] ^ prev) & maxIndex; //(USE THIS IF nRows IS A POWER OF 2)
            rowa = ((unsigned int)state[0] ^ prev) % nRows;		//(USE THIS FOR THE "GENERIC" CASE)

            MemMatrixDev_P = &MemMatrix[(prev * ROW_LEN_INT64)];
            MemMatrixDev_A = &MemMatrix[(rowa * ROW_LEN_INT64)];   
            MemMatrixDev_R = &MemMatrix[(row  * ROW_LEN_INT64)];  
            
            //Performs a reduced-round duplexing operation over M[row*] XOR M[prev], updating both M[row*] and M[row]
            reducedDuplexRow2(state, MemMatrixDev_P , MemMatrixDev_A, MemMatrixDev_R);
            
            //Goes to the next row (direct order)
            prev = row;
            row++;
        } while (row <= maxIndex);	
	}
	*rowA = rowa;
}

/**
 Prints an array of unsigned chars
 */
void printArray(unsigned char *array, unsigned int size, char *name) {
    int i;
    printf("%s: ", name);
    for (i = 0; i < size; i++) {
        printf("%2x|", array[i]);
    }
    printf("\n");
}
////////////////////////////////////////////////////////////////////////////////////////////////
