#include "hip/hip_runtime.h"
/**
 * A simple attack against Lyra2 Password Hashing Scheme (PHS).
 * This is a specific implementation, used only to start
 * evaluating GPU attacks. This implementation needs improvement
 * in specific GPU optimization technics.
 *
 * Author: The Lyra PHC team (http://www.lyra2.net/) -- 2015.
 *
 * This software is hereby placed in the public domain.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>

#include "attackLyra2.h"
#include "attackSponge.h"

int gpuMult(void *K, unsigned int kLen, unsigned char **passwords, unsigned int pwdlen, unsigned char **salts, unsigned int saltlen, unsigned int timeCost, unsigned int nRows, unsigned int nCols, unsigned int totalPasswords, unsigned int gridSize, unsigned int blockSize) {
    int result = 0;

    //============================= Basic variables ============================//
#if (nPARALLEL > 1)
    int64_t i, j, k; //auxiliary iteration counter
#endif   // nPARALLEL > 1

    hipError_t errorCUDA;
    uint64_t sizeSlice = nRows / nPARALLEL;
    //==========================================================================/
    uint64_t nBlocksInput;

    //Checks kernel geometry configuration
    if ((gridSize * blockSize) != (totalPasswords * nPARALLEL)) {
        printf("Error in thread geometry: (gridSize * blockSize) != (totalPasswords * nPARALLEL).\n");
        return -1;
    }
    //Checks whether or not the salt+password are within the accepted limits
    if (pwdlen + saltlen > ROW_LEN_BYTES) {
        return -1;
    }

    //========== Initializing the Memory Matrix and Keys =============//
    //Allocates the keys
    unsigned char *pKeys = (unsigned char *) malloc(totalPasswords * nPARALLEL * kLen * sizeof (unsigned char));
    if (pKeys == NULL) {
        return -1;
    }

    // GPU memory matrix alloc:
    // Memory matrix: nRows of nCols blocks, each block having BLOCK_LEN_INT64 64-bit words
    uint64_t *memMatrixGPU;
    errorCUDA = hipMalloc((void**) &memMatrixGPU, totalPasswords * nRows * ROW_LEN_BYTES);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Allocates the GPU keys
    unsigned char *pkeysGPU;
    errorCUDA = hipMalloc((void**) &pkeysGPU, totalPasswords * nPARALLEL * kLen * sizeof (unsigned char));
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Sponge state: 16 uint64_t, BLOCK_LEN_INT64 words of them for the bitrate (b) and the remainder for the capacity (c)
    uint64_t *stateThreadGPU;
    errorCUDA = hipMalloc((void**) &stateThreadGPU, totalPasswords * nPARALLEL * STATESIZE_BYTES);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // stateThreadGPU cleanup:
    hipMemset(stateThreadGPU, 0, totalPasswords * nPARALLEL * STATESIZE_BYTES);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory setting error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        return -2;
    }

    //Allocates the State Index to be absorbed by each thread.
    uint64_t *stateIdxGPU;
    errorCUDA = hipMalloc((void**) &stateIdxGPU, totalPasswords * nPARALLEL * BLOCK_LEN_BLAKE2_SAFE_BYTES);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Allocates the Password in GPU.
    unsigned char *pwdGPU;
    errorCUDA = hipMalloc((void**) &pwdGPU, totalPasswords * pwdlen);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // Transfers the password to GPU.
    errorCUDA = hipMemcpy(pwdGPU, passwords[0], totalPasswords * pwdlen, hipMemcpyHostToDevice);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Allocates the Salt in GPU.
    unsigned char *saltGPU;
    errorCUDA = hipMalloc((void**) &saltGPU, totalPasswords * saltlen);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // Transfers the salt to GPU.
    errorCUDA = hipMemcpy(saltGPU, salts[0], totalPasswords * saltlen, hipMemcpyHostToDevice);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //========================== BootStrapping Phase ==========================//
#if (nPARALLEL == 1)
    nBlocksInput = ((saltlen + pwdlen + 6 * sizeof (int)) / BLOCK_LEN_BLAKE2_SAFE_BYTES) + 1;
#endif  // nPARALLEL == 1

#if (nPARALLEL > 1)
    nBlocksInput = ((saltlen + pwdlen + 8 * sizeof (int)) / BLOCK_LEN_BLAKE2_SAFE_BYTES) + 1;
#endif   // nPARALLEL > 1

    bootStrapGPU <<<gridSize, blockSize>>>(memMatrixGPU, pkeysGPU, kLen, pwdGPU, pwdlen, saltGPU, saltlen, timeCost, nRows, nCols, nBlocksInput, totalPasswords);

    // Needs to wait all threads:
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //============== Initializing the Sponge State =============/
    initState <<<gridSize, blockSize>>>(stateThreadGPU, totalPasswords);

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //============= Absorbing the input data with the sponge ===============//
    absorbInput <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, stateIdxGPU, nBlocksInput, totalPasswords);

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //================================ Setup and Wandering Phase =============================//
    //Initializes M[0]
    reducedSqueezeRow0 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, totalPasswords); //The locally copied password is most likely overwritten here

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Initializes M[1]
    reducedDuplexRow1and2 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, totalPasswords, 0, 1);

    // Wait all threads to verify execution errors.
    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Initializes M[2]
    reducedDuplexRow1and2 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, totalPasswords, 1, 2);

    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

#if (nPARALLEL == 1)
    // Runs Setup and Wandering Phase
    setupPhaseWanderingGPU_P1 <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, sizeSlice, totalPasswords, timeCost);
#endif //nParallel == 1

#if (nPARALLEL > 1)
    // Runs Setup and Wandering Phase
    setupPhaseWanderingGPU <<<gridSize, blockSize>>>(memMatrixGPU, stateThreadGPU, sizeSlice, totalPasswords, timeCost);
#endif //nParallel > 1

    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error after SetupWandering: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    //Squeezes the keys
    squeeze <<<gridSize, blockSize>>>(stateThreadGPU, pkeysGPU, kLen, totalPasswords);

    hipDeviceSynchronize();

    errorCUDA = hipGetLastError();
    if (hipSuccess != errorCUDA) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }

    // Getting the keys back.
    errorCUDA = hipMemcpy(pKeys, pkeysGPU, totalPasswords * nPARALLEL * kLen * sizeof (unsigned char), hipMemcpyDeviceToHost);
    if (hipSuccess != errorCUDA) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(errorCUDA));
        return -2;
    }


#if (nPARALLEL > 1)
    // XORs all Keys
    for (k = 0; k < totalPasswords; k++) {
        for (i = 1; i < nPARALLEL; i++) {
            for (j = 0; j < kLen; j++) {
                pKeys[k * kLen * nPARALLEL + j] ^= pKeys[k * kLen * nPARALLEL + i * kLen + j];
            }
        }
    }

    //Move the keys to proper place
    for (k = 1; k < totalPasswords; k++) {
        for (j = 0; j < kLen; j++) {
            pKeys[k * kLen + j] = pKeys[k * kLen * nPARALLEL + j];
        }
    }
#endif //nParallel > 1

    // Returns in the correct variable
    memcpy(K, pKeys, totalPasswords * kLen * sizeof (unsigned char));

    //========== Frees the Memory Matrix and Keys =============//
    hipFree(memMatrixGPU);
    hipFree(pkeysGPU);
    hipFree(stateThreadGPU);
    hipFree(stateIdxGPU);
    hipFree(saltGPU);
    hipFree(pwdGPU);

    //Free allKeys
    free(pKeys);
    pKeys = NULL;

    return result;
}
