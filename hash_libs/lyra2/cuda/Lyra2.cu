#include "hip/hip_runtime.h"
/**
 * Implementation of the Lyra2 Password Hashing Scheme (PHS). 
 * Experimental CUDA implementation.
 * 
 * Note: Implemented without shared memory optimizations.
 * 
 * Author: The Lyra PHC team (http://www.lyra-kdf.net/) -- 2014.
 * 
 * This software is hereby placed in the public domain.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHORS ''AS IS'' AND ANY EXPRESS
 * OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHORS OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
 * BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
 * WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
 * OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "Lyra2.h"
#include "Sponge.h"


/**
 * Executes Lyra2 based on the G function from Blake2b. The number of columns of the memory matrix is set to nCols = 64.
 * This version supports salts and passwords whose combined length is smaller than the size of the memory matrix, 
 * (i.e., (nRows x nCols x b) bits,  where "b" is the underlying sponge's bitrate) 
 * 
 * @param out     The derived key to be output by the algorithm
 * @param outlen  Desired key length
 * @param in      User password
 * @param inlen   Password length
 * @param salt    Salt
 * @param saltlen Salt length
 * @param t_cost  Parameter to determine the processing time (T)
 * @param m_cost  Memory cost parameter (defines the number of rows of the memory matrix, R)
 * 
 * @return          0 if the key is generated correctly; -1 if there is an error (usually due to lack of memory for allocation)
 */
int PHS(void *out, size_t outlen, const void *in, size_t inlen, const void *salt, size_t saltlen, unsigned int t_cost, unsigned int m_cost){
	const unsigned char *inPWD = (const unsigned char *)in;
	const unsigned char *saltG = (const unsigned char *)salt;
	unsigned char *outK = (unsigned char *)out;

	return LYRA2(outK, outlen, inPWD, inlen, saltG, saltlen, t_cost, m_cost, N_COLS);
}


void print64(uint64_t *v){
    int i;
    for (i = 0; i < 16; i++)    {
        printf("%16lx|",v[i]);
    }
    printf("\n");
}
/**
 * Executes Lyra2 based on the G function from Blake2b. This version supports salts and passwords
 * whose combined length is smaller than the size of the memory matrix, (i.e., (nRows x nCols x b) bits, 
 * where "b" is the underlying sponge's bitrate) 
 * 
 * @param K         The derived key to be output by the algorithm
 * @param kLen      Desired key length
 * @param pwd       User password
 * @param pwdlen    Password length
 * @param salt      Salt
 * @param saltlen   Salt length
 * @param timeCost  Parameter to determine the processing time (T)
 * @param nRows     Number or rows of the memory matrix (R)
 * @param nCols     Number of columns of the memory matrix (C)
 * 
 * @return          0 if the key is generated correctly; -1 if there is an error (usually due to lack of memory for allocation)
 */
int LYRA2(unsigned char *K, int kLen, const unsigned char *pwd, int pwdlen, const unsigned char *salt, int saltlen, int timeCost, int nRows, int nCols) {

    int rowaCPU, i;

    //Checks whether or not the salt+password are within the accepted limits
    if (pwdlen + saltlen > ROW_LEN_BYTES) {
        return -1;
    }

    // GPU memory matrix alloc:
    // Memory matrix: nRows of nCols blocks, each block having BLOCK_LEN_INT64 64-bit words
    uint64_t *MemMatrixDev;
    hipMalloc((void***) &MemMatrixDev, nRows * ROW_LEN_BYTES);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        exit(EXIT_FAILURE);
    }

    // Memory matrix cleanup:
    hipMemset(MemMatrixDev, 0, nRows * ROW_LEN_BYTES);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory setting error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        exit(EXIT_FAILURE);
    }

    // CPU state alloc:
    //Sponge state (initialized to zeros): 16 uint64_t, 8 of them for the bitrate (b) and the remainder 8 for the capacity (c)
    uint64_t *stateHost = (uint64_t *) malloc(16 * sizeof (uint64_t));
    if (stateHost == NULL) {
        printf("Malloc error in file %s, line %d!\n", __FILE__, __LINE__);
        hipFree(MemMatrixDev);
        free(stateHost);
        MemMatrixDev = NULL;
        exit(EXIT_FAILURE);
    }
    memset(stateHost, 0, 16 * sizeof (uint64_t));

    // GPU state alloc:
    uint64_t *stateDev;
    hipMalloc((void**) &stateDev, 16 * sizeof (uint64_t));
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        free(stateHost);
        hipFree(stateDev);
        stateDev = NULL;
        exit(EXIT_FAILURE);
    }
    
    // GPU state cleanup
    hipMemset(stateDev, 0, 16 * sizeof (uint64_t));
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory setting error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        exit(EXIT_FAILURE);
    }

    // GPU rowa alloc:
    int *rowADev;
    hipMalloc((void**) &rowADev, sizeof (int));
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory allocation error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }

    hipMemset(rowADev, 0, sizeof (int));
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory setting error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }


    //============== Initialing the Sponge State =============//
    initState(stateDev);
    //========================================================//

    //============= Getting the password + salt padded with 10*1 ===============//
    uint64_t * MemMatrixHost = (uint64_t*) malloc(ROW_LEN_BYTES);
    if (MemMatrixHost == NULL) {
        printf("Malloc error in file %s, line %d!\n", __FILE__, __LINE__);
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        free(MemMatrixHost);
        exit(EXIT_FAILURE);
    }
    memset(MemMatrixHost, 0, ROW_LEN_BYTES);

    //Prepends the salt to the password    
    byte *ptrMem = (byte*) MemMatrixHost;
    memcpy(ptrMem, salt, saltlen);

    //Concatenates the password
    ptrMem += saltlen;
    memcpy(ptrMem, pwd, pwdlen);

    //Now comes the padding
    ptrMem += pwdlen;
    *ptrMem = 0x80; //first byte of padding: right after the password

    //Computes the number of blocks taken by the salt and password (from 1 to nCols)
    int nBlocksInput = ((saltlen + pwdlen) / BLOCK_LEN_BYTES) + 1;
    ptrMem = (byte*) (MemMatrixHost);
    ptrMem += nBlocksInput * BLOCK_LEN_BYTES - 1; //sets the pointer to the correct position: end of incomplete block
    *ptrMem ^= 0x01; //last byte of padding: at the end of the last incomplete block

    //Copy the result to GPU memory
    hipMemcpy(MemMatrixDev, MemMatrixHost, ROW_LEN_BYTES, hipMemcpyHostToDevice);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }
    //Clean local password 
    memset(MemMatrixHost, 0, ROW_LEN_BYTES);
    //========================================================//

    //====================== Setup Phase =====================//
    ////Absorbing salt and password
    uint64_t *ptrWord = MemMatrixDev;
    for (i = 0; i < nBlocksInput; i++) {
        absorbBlock << <1, 1 >> >(stateDev, ptrWord); //absorbs each block of pad(salt || pwd)
        if (hipSuccess != hipGetLastError()) {
            printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
            printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
            free(stateHost);
            hipFree(stateDev);
            hipFree(MemMatrixDev);
            MemMatrixDev = NULL;
            stateDev = NULL;
            hipFree(rowADev);
            rowADev = NULL;
            exit(EXIT_FAILURE);
        }
        ptrWord = &MemMatrixDev[((i + 1) * BLOCK_LEN_INT64)]; //goes to next block of pad(salt || pwd)
    }
    //========================================================//

    //Initializes M[0] and M[1]
    reducedSqueezeRow << <1, 1 >> >(stateDev, MemMatrixDev); //The GPU copied password is overwritten here
    ptrWord = &MemMatrixDev[(ROW_LEN_INT64)];
    reducedSqueezeRow << <1, 1 >> >(stateDev, ptrWord);

    setupGPU << <1, 1 >> >(stateDev, MemMatrixDev, nRows);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }

    //================== Wandering Phase =====================//
    wandering << <1, 1 >> > (stateDev, MemMatrixDev, timeCost, nRows, rowADev);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }

    //Recover rowa from GPU
    hipMemcpy(&rowaCPU, rowADev, sizeof (int), hipMemcpyDeviceToHost);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA memory copy error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }
    //========================================================//

    //==================== Wrap-up Phase =====================//
    //Absorbs
    absorbBlock << <1, 1 >> >(stateDev, &MemMatrixDev[(rowaCPU * ROW_LEN_INT64)]);
    if (hipSuccess != hipGetLastError()) {
        printf("CUDA kernel call error in file %s, line %d!\n", __FILE__, __LINE__);
        printf("Error: %s \n", hipGetErrorString(hipGetLastError()));
        free(stateHost);
        hipFree(stateDev);
        hipFree(MemMatrixDev);
        MemMatrixDev = NULL;
        stateDev = NULL;
        hipFree(rowADev);
        rowADev = NULL;
        exit(EXIT_FAILURE);
    }


    //Squeezes the key
    squeeze(stateDev, K, kLen);
    //========================================================//

    //=============== Freeing the memory =====================//
    hipFree(MemMatrixDev);
    hipFree(stateDev);
    hipFree(rowADev);
    free(stateHost);
    free(MemMatrixHost);
    MemMatrixDev = NULL;
    stateDev = NULL;
    rowADev = NULL;
    stateHost = NULL;
    MemMatrixHost = NULL;
    //========================================================//

    return 0;
}




